#include "hip/hip_runtime.h"
#include <torch/library.h>

#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/util/Exception.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

#if defined(TORCHAO_USE_CUTLASS) && !defined(_WIN32) &&                   \
    defined(CUDA_VERSION) && (CUDA_VERSION >= 12080)
#define BUILD_MX_KERNELS_CUTLASS
#endif

#if defined(BUILD_MX_KERNELS_CUTLASS)

#include "cute/tensor.hpp"
#include "cutlass/detail/sm100_blockscaled_layout.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/util/packed_stride.hpp"


#endif

namespace torchao {

#if defined(BUILD_MX_KERNELS_CUTLASS)
namespace {

using namespace cute;

template<typename Element>
constexpr int GetAlignment() {
    if constexpr (std::is_same_v<Element, cutlass::mx_float4_t<cutlass::float_e2m1_t>>)
        return 32;
    return 16;
}

template <typename ElementA,
          typename ElementB,
          typename ElementD,
          typename MmaTileShape,
          typename ClusterShape,
          typename PerSmTileShape_MNK>
void run_gemm(at::Tensor& a, at::Tensor& b, at::Tensor& a_scale,
             at::Tensor& b_scale, at::Tensor& out, int M, int K, int N) {
  // A matrix configuration
  using         LayoutATag  = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
  constexpr int AlignmentA  = GetAlignment<ElementA>();    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

  // B matrix configuration
  using         LayoutBTag  = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
  constexpr int AlignmentB  = GetAlignment<ElementB>();    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

  // C/D matrix configuration
  using         ElementC    = cutlass::bfloat16_t;                            // Element type for C matrix operand
  using         LayoutCTag  = cutlass::layout::RowMajor;                      // Layout type for C matrix operand
  using         LayoutDTag  = cutlass::layout::RowMajor;                      // Layout type for D matrix operand
  constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;    // Memory access granularity/alignment of D matrix in units of elements (up to 16 bytes)
  constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)
  // Kernel functional config
  using ElementAccumulator  = float;                                          // Element type for internal accumulation
  using ArchTag             = cutlass::arch::Sm100;                           // Tag indicating the minimum SM that supports the intended feature
  using OperatorClass       = cutlass::arch::OpClassBlockScaledTensorOp;      // Operator class tag


  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      PerSmTileShape_MNK, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementAccumulator,
      ElementC, LayoutCTag, AlignmentC,
      ElementD, LayoutDTag, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto                      // Epilogue schedule policy
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      ElementA, LayoutATag, AlignmentA,
      ElementB, LayoutBTag, AlignmentB,
      ElementAccumulator,
      MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto                             // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,                                                   // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue,
      void>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // Reference device GEMM implementation type
  using StrideA   = typename Gemm::GemmKernel::StrideA;
  using StrideB   = typename Gemm::GemmKernel::StrideB;
  using StrideC   = typename Gemm::GemmKernel::StrideC;
  using StrideD   = typename Gemm::GemmKernel::StrideD;
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using Sm100BlkScaledConfig = typename Gemm::GemmKernel::CollectiveMainloop::Sm100BlkScaledConfig;

  // Initialize strides using packed stride configuration
  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, make_shape(M, K, 1));
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, make_shape(N, K, 1));
  auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, make_shape(M, N, 1));

  // Initialize scale factor layouts using block scaled configuration
  auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(make_shape(M, N, K, 1));
  auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(make_shape(M, N, K, 1));

  using DtypeA = typename ElementA::DataType;
  using DtypeB = typename ElementB::DataType;
  using DtypeScaleA = typename ElementA::ScaleFactorType;
  using DtypeScaleB = typename ElementB::ScaleFactorType;
  using DtypeOut = ElementD;

  Gemm gemm;

  auto A_ptr = reinterpret_cast<DtypeA*>(a.data_ptr());
  auto B_ptr = reinterpret_cast<DtypeB*>(b.data_ptr());
  auto SFA_ptr = reinterpret_cast<DtypeScaleA*>(a_scale.data_ptr());
  auto SFB_ptr = reinterpret_cast<DtypeScaleB*>(b_scale.data_ptr());
  auto out_ptr = reinterpret_cast<DtypeOut*>(out.data_ptr());

  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {M, N, K, 1},
    { // Mainloop arguments
      A_ptr, stride_A,
      B_ptr, stride_B,
      SFA_ptr, layout_SFA,
      SFB_ptr, layout_SFB
    },
    { // Epilogue arguments
      {1.0, 0.0},
      nullptr, StrideC{},  // No bias for now
      out_ptr, stride_D
    }
  };

  // arguments.scheduler.max_swizzle_size = 8;

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  TORCH_CHECK(status == cutlass::Status::kSuccess, "Cutlass cannot implement");
  // Allocate workspace memory
  size_t workspace_size = Gemm::get_workspace_size(arguments);
  auto workspace = a.new_empty(
      {static_cast<int64_t>(workspace_size)},
      at::TensorOptions().dtype(at::kByte));


  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.data_ptr());
  TORCH_CHECK(status == cutlass::Status::kSuccess, "Cutlass cannot initialize");

  status = gemm.run(at::cuda::getCurrentCUDAStream());
  TORCH_CHECK(status == cutlass::Status::kSuccess, "Cutlass cannot run", cutlass::cutlassGetStatusString(status));

  C10_CUDA_KERNEL_LAUNCH_CHECK();

}
}
#endif

void validate(at::Tensor a, at::Tensor b, at::Tensor a_scale, at::Tensor b_scale){
    TORCH_CHECK(a.is_cuda(), "a must be CUDA tensor");
    TORCH_CHECK(b.is_cuda(), "b must be CUDA tensor");
    TORCH_CHECK(a_scale.is_cuda(), "a_scale must be CUDA tensor");
    TORCH_CHECK(b_scale.is_cuda(), "b_scale must be CUDA tensor");

    // Check matrix dimensions
    TORCH_CHECK(a.dim() == 2, "a must be a matrix");
    TORCH_CHECK(b.dim() == 2, "b must be a matrix");

    // Get dimensions
    auto M = a.size(0);
    auto K = a.size(1);
    auto N = b.size(1);

    TORCH_CHECK(b.size(0) == K,
        "Incompatible matrix dimensions: a is ", M, "x", K, " but b is ", b.size(0), "x", N);

    // Needed for TMA store
    TORCH_CHECK(N % 8 == 0, "N must be a multiple of 16 but got, ", N);

    // Check 16-byte alignment for input tensors
    TORCH_CHECK(
        reinterpret_cast<std::uintptr_t>(a.data_ptr()) % 16 == 0,
        "Input tensor 'a' must be 16-byte aligned");
    TORCH_CHECK(
        reinterpret_cast<std::uintptr_t>(b.data_ptr()) % 16 == 0,
        "Input tensor 'b' must be 16-byte aligned");

    auto ceil_div = [](auto a, auto b) { return (a + b - 1) / b; };
    auto num_k_blocks = ceil_div(K, 32);
    // For a_scale, we expect elements or M* ceil(K/32) elements
    auto expected_a_scale_size = 128 * ceil_div(M, 128) * num_k_blocks;
    TORCH_CHECK(a_scale.numel() == expected_a_scale_size, "Expected b_scale_size to be ", expected_a_scale_size, " but got ", a_scale.numel());

    // For b_scale, we expect N * ceil(K/32) elements
    auto expected_b_scale_size = 128 * ceil_div(N, 128) * num_k_blocks;
    TORCH_CHECK(b_scale.numel() == expected_b_scale_size, "Expected a_scale_size to be ", expected_b_scale_size, " but got ", b_scale.numel());

    // Check tensor strides for optimal memory layout
    TORCH_CHECK(
        a.stride(1) == 1,
        "Input tensor 'a' must be contiguous in the K dimension (row-major)");
    TORCH_CHECK(
        b.stride(0) == 1,
        "Input tensor 'b' must be contiguous in the K dimension (column-major)");
}


at::Tensor mx_fp8_bf16(at::Tensor a, at::Tensor b, at::Tensor a_scale,
                       at::Tensor b_scale) {
#if defined(BUILD_MX_KERNELS_CUTLASS)
  validate(a, b, a_scale, b_scale);
  auto M = a.size(0);
  auto K = a.size(1);
  auto N = b.size(1);

  auto out =
      at::empty({M, N}, a.options().dtype(at::kBFloat16));
  using ElementA = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
  using ElementD = cutlass::bfloat16_t;

  using MmaTileShape        = Shape<_128,_128,_128>;
  using ClusterShape        = Shape<_2,_1,_1>;
  using PerSmTileShape_MNK  = Shape<_128,_128,_128>;

  run_gemm<ElementA, ElementB, ElementD, MmaTileShape, ClusterShape, PerSmTileShape_MNK>(a, b, a_scale, b_scale, out, M, K, N);
  return out;
  #else
  TORCH_CHECK_NOT_IMPLEMENTED(false, __func__);
  return at::Tensor{};
#endif
}

at::Tensor mx_fp4_bf16(at::Tensor a, at::Tensor b, at::Tensor a_scale,
                       at::Tensor b_scale) {
#if defined(BUILD_MX_KERNELS_CUTLASS)
  TORCH_CHECK(a.is_cuda(), "a must be CUDA tensor");
  TORCH_CHECK(b.is_cuda(), "b must be CUDA tensor");
  TORCH_CHECK(a_scale.is_cuda(), "a_scale must be CUDA tensor");
  TORCH_CHECK(b_scale.is_cuda(), "b_scale must be CUDA tensor");

  auto M = a.size(0);
  auto K = a.size(1) * 2;
  auto N = b.size(1);

  auto out =
      at::empty({M, N}, a.options().dtype(at::kBFloat16));
  using ElementA = cutlass::mx_float4_t<cutlass::float_e2m1_t>;
  using ElementB = cutlass::mx_float4_t<cutlass::float_e2m1_t>;
  using ElementD = cutlass::bfloat16_t;

  using MmaTileShape        = Shape<_128,_128,_128>;
  using ClusterShape        = Shape<_2,_1,_1>;
  using PerSmTileShape_MNK  = Shape<_128,_128,_128>;

  run_gemm<ElementA, ElementB, ElementD, MmaTileShape, ClusterShape, PerSmTileShape_MNK>(a, b, a_scale, b_scale, out, M, K, N);
  return out;
#else
  TORCH_CHECK_NOT_IMPLEMENTED(false, __func__);
  return at::Tensor{};
#endif
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::mx_fp8_bf16", &mx_fp8_bf16);
}
TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::mx_fp4_bf16", &mx_fp4_bf16);
}



} // namespace torchao

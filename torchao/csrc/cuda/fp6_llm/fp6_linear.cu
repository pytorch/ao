#include "hip/hip_runtime.h"
//    Copyright 2024 FP6-LLM authors
//
//    Licensed under the Apache License, Version 2.0 (the "License");
//    you may not use this file except in compliance with the License.
//    You may obtain a copy of the License at
//
//        http://www.apache.org/licenses/LICENSE-2.0
//
//    Unless required by applicable law or agreed to in writing, software
//    distributed under the License is distributed on an "AS IS" BASIS,
//    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//    See the License for the specific language governing permissions and
//    limitations under the License.
// 
// This file is adapted from https://github.com/usyd-fsalab/fp6_llm/blob/5df6737cca32f604e957e3f63f03ccc2e4d1df0d/fp6_llm/csrc/fp6_linear.cu

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 800  // at least Ampere

#include "kernel_matmul.cuh"
#include "kernel_reduction.cuh"

#include <stdio.h>
#include <assert.h>

template<typename TilingConfig, typename OutputDataType, int EXPONENT, int MANTISSA>
static void Kernel_Ex(hipStream_t    stream,
                      const uint4     *Weight,
                      const half      *Scales,
                      const half      *B,
                      OutputDataType  *C,
                      const size_t    M_Global,
                      const size_t    N_Global,
                      const size_t    K_Global, 
                      int             Split_K) 
{
    #ifdef DEBUG_MODE
        printf("\n");
        printf("Launcher.cu->Kernel_Ex():\n");
        printf("M: %d, N: %d, K: %d, SplitK: %d\n", M_Global, N_Global, K_Global, Split_K);
        printf("TILE_M: %d, TILE_K: %d, TILE_N: %d\n", TilingConfig::TILE_M, TilingConfig::TILE_K, TilingConfig::TILE_N);
    #endif
    static size_t SHMEM_SZ = max(TilingConfig::SMEM_SIZE_B_TILE+SMEM_SIZE_PER_TB_A_TILE, TilingConfig::SMEM_SIZE_C_TILE);
    hipFuncSetAttribute(reinterpret_cast<const void*>(QUANT_GEMM_Kernel<TilingConfig), OutputDataType, EXPONENT, MANTISSA>, hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ);
    size_t  dimN = (N_Global-1) / TilingConfig::TILE_N + 1;
    size_t  dimM = M_Global * Split_K / TilingConfig::TILE_M;
    dim3    GridDim(dimN, dimM, 1);
    dim3    BlockDim(WARP_SIZE * TilingConfig::BLOCK_WARPS, 1, 1);
    //
    #ifdef DEBUG_MODE
        printf("GridDim.x: %d, GridDim.y: %d, GridDim.z: %d, BlockDim.x: %d, BlockDim.y: %d, BlockDim.z: %d SHMEM_SZ: %d\n",
                GridDim.x, GridDim.y, GridDim.z, BlockDim.x, BlockDim.y, BlockDim.z, SHMEM_SZ);
        printf("\n");
    #endif
    QUANT_GEMM_Kernel<TilingConfig, OutputDataType, EXPONENT, MANTISSA><<<GridDim, BlockDim, SHMEM_SZ, stream>>>
                    (Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);
}

template<int EXPONENT, int MANTISSA>
hipError_t fpx_linear_kernel(hipStream_t    stream,
                              const uint4     *Weight,
                              const half      *Scales,
                              const half      *B,
                              half            *C,
                              const size_t    M_Global,
                              const size_t    N_Global,
                              const size_t    K_Global, 
                              float           *Reduction_Workspace,  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
                              int             Split_K)
{
    assert(M_Global % 256 == 0);
    assert(K_Global % 64 == 0);
    assert(N_Global>0);

    // Work around to support more N shapes:
    size_t N_PowerOf2;
    if(N_Global>0 &&  N_Global<=8)      N_PowerOf2 = 8;
    if(N_Global>8 &&  N_Global<=16)     N_PowerOf2 = 16;
    if(N_Global>16 && N_Global<=32)     N_PowerOf2 = 32;
    if(N_Global>32 && N_Global<=64)     N_PowerOf2 = 64;
    if(N_Global>64 && N_Global<=128)    N_PowerOf2 = 128;
    if(N_Global>128)                    N_PowerOf2 = ((N_Global-1)/128+1) * 128;

    if (Split_K == 1) {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
        }
    }
    else {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
        }
        // Reduction for SplitK
        dim3 GridDim((M_Global * N_Global) / REDUCTION_ELEMENT_PER_THREADBLOCK, 1, 1);
        dim3 BlockDim(WARP_SIZE, 1, 1);
        SplitK_Reduction<<<GridDim, BlockDim, 0, stream>>>(C, Reduction_Workspace, M_Global, N_Global, Split_K);
    }
    return hipGetLastError();
}


#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/library.h>

namespace torchao {
// MODIFICATION NOTE: dtype of _weights is changed to uint8
/*
Computes FPx-FP16 GEMM (PyTorch interface).

[Mathmatical Formula]
Standard definition of linear layer:    Out = In * trans(W), where In, Out, and W are stored in row-major.
After Equivalent transformation    :    trans(Out) = W * trans(In). Note that we do not perform "transpose" during runtime, we instead interpret the In/Out as column-major matrices when calling our CUDA kernel.

[Inputs]
  _in_feats:  tensor of shape [B, IC];                  // half 
  _weights:   int tensor of shape [OC, IC // 8 * x];    // x UINT8 words contains 8 FPx weights.
  _scales:    tensor of shape [OC];                     // half
  splitK:     spliting the MatMul problem along K dimension for higher GPU utilization, default 1.
[Outputs]
  _out_feats: tensor of shape [B, OC];                  // half
*/
torch::Tensor fp_eXmY_linear_forward_cuda(
    int64_t         EXPONENT,
    int64_t         MANTISSA,
    torch::Tensor   _in_feats,
    torch::Tensor   _weights,
    torch::Tensor   _scales,
    int64_t         splitK=1)
{
    const int64_t NBITS   = 1 + EXPONENT + MANTISSA;
    int num_in_feats      = _in_feats.size(0);
    int num_in_channels   = _in_feats.size(1);
    int num_out_channels  = _weights.size(0);
    TORCH_CHECK(num_in_channels % 64 == 0, "Expected in_features to be a multiple of 64, but received ", num_in_channels);
    TORCH_CHECK((num_in_channels / 8 * NBITS) == _weights.size(1));    // Making sure the K dimension is matched.
    //
    int M = num_out_channels;
    int K = num_in_channels;
    int N = num_in_feats;
    // Input Tensors
    auto weight = reinterpret_cast<const uint4*>(_weights.data_ptr<uint8_t>());  // weights is [OC, IC] but in FP6.
    auto in_feats = reinterpret_cast<const half*>(_in_feats.data_ptr<at::Half>());
    auto scales   = reinterpret_cast<const half*>(_scales.data_ptr<at::Half>());
    // Output Tensors
    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty({num_in_feats, num_out_channels}, options);
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());

    options = torch::TensorOptions().dtype(torch::kFloat32).device(_in_feats.device());
    at::Tensor _workspace = torch::empty({splitK, num_in_feats, num_out_channels}, options);
    auto Reduction_Workspace = reinterpret_cast<float*>(_workspace.data_ptr<float>());  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)

    // MODIFICATION NOTE: use at::cuda::getCurrentCUDAStream() instead of default stream (0)
    // this fixes problem with CUDA graphs when used with torch.compile()
    auto stream = at::cuda::getCurrentCUDAStream();

    // officially supported in Quant-LLM
    if (EXPONENT == 3 && MANTISSA == 2)
        fpx_linear_kernel<3, 2>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);
    else if (EXPONENT == 2 && MANTISSA == 2)
        fpx_linear_kernel<2, 2>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);

    // experimental
    else if (EXPONENT == 2 && MANTISSA == 3)
        fpx_linear_kernel<2, 3>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);
    else if (EXPONENT == 3 && MANTISSA == 1)
        fpx_linear_kernel<3, 1>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);
    // else if (EXPONENT == 2 && MANTISSA == 1)
    //     fpx_linear_kernel<2, 1>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);
    // else if (EXPONENT == 3 && MANTISSA == 0)
    //     fpx_linear_kernel<3, 0>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);
    // else if (EXPONENT == 2 && MANTISSA == 0)
    //     fpx_linear_kernel<2, 0>(stream, weight, scales, in_feats, out_feats, M, N, K, Reduction_Workspace, splitK);

    else
        TORCH_CHECK(false, "FP", NBITS, " E", EXPONENT, "M", MANTISSA, " is not supported.");

    return _out_feats;
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::quant_llm_linear", &fp_eXmY_linear_forward_cuda);
}

} // namespace torchao

#endif
